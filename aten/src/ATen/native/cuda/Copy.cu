#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/Dispatch_v2.h>
#include <ATen/cuda/CachingHostAllocator.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAEvent.h>
#include <ATen/cuda/PeerToPeerAccess.h>
#include <ATen/native/Copy.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#else
#include <ATen/ops/empty_like.h>
#endif

#include <c10/cuda/CUDACachingAllocator.h>
#include <c10/cuda/CUDAStream.h>

// TODO(NS): Investigate why FP8 conversion intrinsics end up being slower
#ifdef AT_USE_NV_CVT_INTRINSICS
#include <hip/hip_fp8.h>
#endif

namespace at::native {

void neg_kernel_cuda(TensorIteratorBase &iter);
void conj_kernel_cuda(TensorIteratorBase &iter);

void float16_copy_kernel_cuda(TensorIteratorBase &iter) {
    gpu_kernel_nocast(iter, [] GPU_LAMBDA(float value) {
        return static_cast<at::Half>(value);
    });
}

void bfloat16_copy_kernel_cuda(TensorIteratorBase &iter) {
    gpu_kernel_nocast(iter, [] GPU_LAMBDA(float value) {
        return static_cast<at::BFloat16>(value);
    });
}

void float8_copy_kernel_cuda(TensorIteratorBase &iter) {
  ScalarType dtype = iter.dtype(0);
  ScalarType other_dtype = iter.dtype(1);
  if (dtype == kFloat8_e4m3fn) {
    switch (other_dtype) {
      case kFloat:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(float value) {
             return Float8_e4m3fn(value);
         });
         break;
      case kHalf:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(Half value) {
             return Float8_e4m3fn(value);
         });
         break;
      case kBFloat16:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(BFloat16 value) {
             return Float8_e4m3fn(value);
         });
         break;
      default:
        gpu_kernel(iter, [] GPU_LAMBDA(Float8_e4m3fn x) { return x; });
        break;
    }
  } else if (dtype == kFloat8_e5m2) {
    switch (other_dtype) {
      case kFloat:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(float value) {
#ifdef AT_USE_NV_CVT_INTRINSICS
             const auto x =  __hip_cvt_float_to_fp8(value, __HIP_NOSAT, __HIP_E5M2_FNUZ);
             return Float8_e5m2(x, Float8_e5m2::from_bits());
#else
             return Float8_e5m2(value);
#endif
         });
         break;
      case kHalf:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(Half value) {
#ifdef AT_USE_NV_CVT_INTRINSICS
             const auto x =  __hip_cvt_halfraw_to_fp8(static_cast<__half>(value), __HIP_NOSAT, __HIP_E5M2_FNUZ);
             return Float8_e5m2(x, Float8_e5m2::from_bits());
#else
             return Float8_e5m2(value);
#endif
         });
         break;
      case kBFloat16:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(BFloat16 value) {
#ifdef AT_USE_NV_CVT_INTRINSICS
             const auto x =  __hip_cvt_bfloat16raw_to_fp8(static_cast<__hip_bfloat16>(value), __HIP_NOSAT, __HIP_E5M2_FNUZ);
             return Float8_e5m2(x, Float8_e5m2::from_bits());
#else
             return Float8_e5m2(value);
#endif
         });
         break;
      default:
         gpu_kernel(iter, [] GPU_LAMBDA(Float8_e5m2 x) { return x; });
         break;
    }
  } else if (dtype == kFloat8_e4m3fnuz) {
    switch (other_dtype) {
      case kFloat:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(float value) {
             return Float8_e4m3fnuz(value);
         });
         break;
      case kHalf:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(Half value) {
             return Float8_e4m3fnuz(value);
         });
         break;
      case kBFloat16:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(BFloat16 value) {
             return Float8_e4m3fnuz(value);
         });
         break;
      default:
        gpu_kernel(iter, [] GPU_LAMBDA(Float8_e4m3fnuz x) { return x; });
        break;
    }
  } else if (dtype == kFloat8_e5m2fnuz) {
    switch (other_dtype) {
      case kFloat:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(float value) {
             return Float8_e5m2fnuz(value);
         });
         break;
      case kHalf:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(Half value) {
             return Float8_e5m2fnuz(value);
         });
         break;
      case kBFloat16:
         gpu_kernel_nocast(iter, [] GPU_LAMBDA(BFloat16 value) {
             return Float8_e5m2fnuz(value);
         });
         break;
      default:
         gpu_kernel(iter, [] GPU_LAMBDA(Float8_e5m2fnuz x) { return x; });
         break;
    }
  } else {
    TORCH_CHECK(false, "This supposed ot be called only for Float8 types");
  }
}

// This API is for detecting whether the permute parameter of a three-dimensional tensor
// in the Copy operation from src to dst is from [0, 1, 2] to [0, 2, 1].
bool is_permute_021(TensorIteratorBase &iter) {
  const auto& input = iter.tensor(1);
  const auto& output = iter.tensor(0);

  bool is_permute = false;
  if (input.dim() == 3) {
    is_permute = true;
    is_permute &= input.dim() == output.dim();
    is_permute &= input.stride(0) == input.size(1) * input.size(2);
    is_permute &= input.stride(1) == 1;
    is_permute &= input.stride(2) == input.size(1);
    is_permute &= output.is_contiguous();
  }
  return is_permute;
}

template<class _T, int _WG>
__global__ void transpose_tile_big_kernel(const void* __restrict a, void* __restrict c, const int N, const int K)
{
    constexpr uint32_t BIG_TILE_SIZE = 64;
    // pad LDS row by dword
    constexpr uint32_t LDS_PAD = (4 / sizeof(_T));
    constexpr uint32_t element_size = sizeof(_T);  // in bytes
    constexpr uint32_t elements_in_16B = 16 / element_size;

    union BLOCK_16B
    {
        _T e[elements_in_16B];
        __uint128_t ow;
    };
    // Round up processing to next full tile
    const uint32_t n_tiles = (N + BIG_TILE_SIZE - 1) / BIG_TILE_SIZE;
    const uint32_t k_tiles = (K + BIG_TILE_SIZE - 1) / BIG_TILE_SIZE;
    const uint32_t nk_tiles = n_tiles * k_tiles;
    const uint32_t m = blockIdx.x / nk_tiles;
    const uint64_t stride_n = N * sizeof(_T);
    const uint64_t stride_k = K * sizeof(_T);
    const uint64_t stride_nk = N * K * sizeof(_T);

    // Walk destination tiles continuously for cache coherency
    constexpr uint32_t XCD = 8;
    constexpr uint32_t SEQ = 8;
    constexpr uint32_t sblk = XCD * SEQ;
    const uint32_t max_swizzle = (nk_tiles / sblk) * sblk;
    uint32_t tIdx = blockIdx.x % nk_tiles;
    tIdx = tIdx > max_swizzle ? tIdx :
        (tIdx / sblk) * sblk + (tIdx % sblk) / SEQ + (tIdx % SEQ) * XCD;
    uint32_t ti = tIdx / k_tiles;
    uint32_t tj = tIdx % k_tiles;

    __shared__ _T sa[BIG_TILE_SIZE][BIG_TILE_SIZE + LDS_PAD];

    // Detect partial tiles
    uint32_t max_part_n = (ti == (n_tiles - 1) && (N % BIG_TILE_SIZE) != 0) ? (N % BIG_TILE_SIZE) : BIG_TILE_SIZE;
    uint32_t max_part_k = (tj == (k_tiles - 1) && (K % BIG_TILE_SIZE) != 0) ? (K % BIG_TILE_SIZE) : BIG_TILE_SIZE;

    if (max_part_n == BIG_TILE_SIZE && max_part_k == BIG_TILE_SIZE)
    {
        // Copy full tile with large loads
        constexpr uint32_t row_bytes = BIG_TILE_SIZE * sizeof(_T);
        constexpr uint32_t vmem_per_row = row_bytes / sizeof(__uint128_t);
        constexpr uint32_t rows_per_wg = _WG / vmem_per_row;
        constexpr uint32_t vmem_per_thread = BIG_TILE_SIZE / rows_per_wg;
        // Make sure WG isn't too large
        static_assert(vmem_per_thread >= 1);

        const uint8_t* pat = (const uint8_t*)a + tj * BIG_TILE_SIZE * stride_n + ti * row_bytes + m * stride_nk;
        #pragma unroll
        for (uint32_t t = 0; t < vmem_per_thread; t++)
        {
            uint32_t col = threadIdx.x % vmem_per_row;
            uint32_t row = threadIdx.x / vmem_per_row + t * rows_per_wg;
            uint64_t offset = row * stride_n + col * sizeof(__uint128_t);
            const __uint128_t* pfa = (const __uint128_t*)(pat + offset);
            BLOCK_16B d;
            d.ow = *pfa;
            #pragma unroll
            for (uint32_t i = 0; i < elements_in_16B; i++)
            {
                sa[row][col * elements_in_16B + i] = d.e[i];
            }
        }
        __syncthreads();

        const uint8_t* pc = (const uint8_t*)c + ti * BIG_TILE_SIZE * stride_k + tj * row_bytes + m * stride_nk;
        #pragma unroll
        for (uint32_t t = 0; t < vmem_per_thread; t++)
        {
            uint32_t col = threadIdx.x % vmem_per_row;
            uint32_t row = threadIdx.x / vmem_per_row + t * rows_per_wg;
            uint64_t offset = row * stride_k + col * sizeof(__uint128_t);
            BLOCK_16B d;
            // Transpose tile on read from LDS
            #pragma unroll
            for (uint32_t i = 0; i < elements_in_16B; i++)
            {
                d.e[i] = sa[col * elements_in_16B + i][row];
            }
            __uint128_t* pfc = (__uint128_t*)(pc + offset);
            *pfc = d.ow;
        }
    }
    else
    {
        // Copy partial tiles with element accesses
        constexpr uint32_t row_bytes = BIG_TILE_SIZE * sizeof(_T);
        constexpr uint32_t vmem_per_row = BIG_TILE_SIZE;
        constexpr uint32_t rows_per_wg = _WG / vmem_per_row;
        constexpr uint32_t vmem_per_thread = BIG_TILE_SIZE / rows_per_wg;
        // Make sure WG isn't too large
        static_assert(vmem_per_thread >= 1);

        const uint8_t* pat = (const uint8_t*)a + tj * BIG_TILE_SIZE * stride_n + ti * row_bytes + m * stride_nk;
        #pragma unroll
        for (uint32_t t = 0; t < vmem_per_thread; t++)
        {
            uint32_t col = threadIdx.x % vmem_per_row;
            uint32_t row = threadIdx.x / vmem_per_row + t * rows_per_wg;
            uint64_t offset = (col < max_part_n && row < max_part_k) ? row * stride_n + col * 2 : 0;
            const uint16_t* pfa = (const uint16_t*)(pat + offset);
            sa[row][col] = *pfa;
        }
        __syncthreads();

        const uint8_t* pc = (const uint8_t*)c + ti * BIG_TILE_SIZE * stride_k + tj * row_bytes + m * stride_nk;
        #pragma unroll
        for (uint32_t t = 0; t < vmem_per_thread; t++)
        {
            uint32_t col = threadIdx.x % vmem_per_row;
            uint32_t row = threadIdx.x / vmem_per_row + t * rows_per_wg;
            if (col < max_part_k && row < max_part_n)
            {
                uint64_t offset = row * stride_k + col * 2;
                uint16_t* pfc = (uint16_t*)(pc + offset);
                *pfc = sa[col][row];
            }
        }
    }
}

void transpose_last2dim(TensorIteratorBase &iter) {
  void* dst = iter.data_ptr(0);
  void* src = iter.data_ptr(1);
  const auto& input = iter.tensor(1);

  int M = input.size(0);
  int N = input.size(1);
  int K = input.size(2);

  auto stream = c10::cuda::getCurrentCUDAStream();
  constexpr uint32_t BIG_TILE_SIZE = 64;
  int big_tile_wg = M * ((N + BIG_TILE_SIZE - 1) / BIG_TILE_SIZE) * ((K + BIG_TILE_SIZE - 1) / BIG_TILE_SIZE);
  const dim3 grid_dim(big_tile_wg, 1, 1);
  const dim3 block_dim(256, 1, 1);
  transpose_tile_big_kernel<uint16_t, 256><<<grid_dim, block_dim, 0, stream>>>(src, dst, N, K);
}

// TODO: We probably can use the opaque type trick to avoid creating duplicate
// kernels for equivalent bit lengths
void direct_copy_kernel_cuda(TensorIteratorBase &iter) {
  ScalarType dtype = iter.dtype(0);

  if (isQIntType(dtype)) {
    AT_DISPATCH_QINT_TYPES(dtype, "copy_", [&] {
      gpu_kernel(iter, [] GPU_LAMBDA(scalar_t x) { return x; });
    });
  } else if (dtype == kFloat8_e5m2 || dtype == kFloat8_e4m3fn || dtype == kFloat8_e5m2fnuz || dtype == kFloat8_e4m3fnuz) {
     float8_copy_kernel_cuda(iter);
  } else if (iter.dtype(1) == kFloat && (dtype == kBFloat16 || dtype == kHalf)) {
     if (dtype == kBFloat16) {
       bfloat16_copy_kernel_cuda(iter);
     } else {
       float16_copy_kernel_cuda(iter);
     }
  } else if (isBitsType(dtype)) {
    TORCH_CHECK(dtype == iter.dtype(1), "copy_() does not support casting "
      "bits types to different bits types. Source dtype is ", iter.dtype(1), "target dtype is ", dtype);
    AT_DISPATCH_BIT_TYPES(dtype, "copy_", [&] {
      gpu_kernel_nocast(iter, [] GPU_LAMBDA(scalar_t x) { return x; });
    });
  } else if (is_permute_021(iter) && (dtype == kBFloat16 || dtype == kHalf)) {
    transpose_last2dim(iter);
  } else {
    AT_DISPATCH_V2(
        dtype, "copy_", AT_WRAP([&] {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t x) { return x; });
    }), AT_EXPAND(AT_ALL_TYPES_AND_COMPLEX), kHalf, kBool, kBFloat16, kComplexHalf, AT_EXPAND(AT_BAREBONES_UNSIGNED_TYPES));
  }
}

void neg_conj_kernel_cuda(TensorIteratorBase &iter) {
  AT_DISPATCH_COMPLEX_TYPES(iter.common_dtype(), "neg_conj_cuda", [&] {
    gpu_kernel(iter, [] GPU_LAMBDA(scalar_t x) { return -std::conj(x); });
  });
}

using namespace at::cuda;

// device-to-device copy, does type conversion
void copy_device_to_device(TensorIterator& iter,
                           bool non_blocking,
                           bool p2p_enabled) {
  int64_t numel = iter.numel();

  // We can memcpy the memory if both tensors have the same type AND both
  // tensors are contiguous after dimension coalescing and reordering.
  bool same_type = iter.dtype(0) == iter.dtype(1);
  bool same_conj = iter.tensor(0).is_conj() == iter.tensor(1).is_conj();
  bool same_neg = iter.tensor(0).is_neg() == iter.tensor(1).is_neg();
  bool memcpy_eligible = same_type && same_conj && same_neg && iter.is_contiguous();

  Device dst_device = iter.device(0);
  Device src_device = iter.device(1);

  CUDAGuard device_guard(src_device);

  // We always perform the copy on the source device, using the current stream
  // on the source device, and we fully synchronize on both src and dst's
  // current streams for completion of the copy. We have to explicitly do this
  // for non-contig copies. This mimics the behavior of cross-device
  // hipMemcpyAsync on the default stream.
  CUDAStream copy_stream = getCurrentCUDAStream(src_device.index());
  if (src_device != dst_device) {
    // This is a cross-device copy on the src current stream and dst current
    // stream. We perform a two-way barrier between both devices' streams
    // before the copy. This ensures that any write-after-write and
    // write-after-read dependencies on the destination side are handled, so
    // that no one is operating on the dst memory when we perform the copy.
    // src waits on dst barrier (src already waits on src)
    CUDAEvent dst_ready;
    device_guard.set_device(dst_device);
    dst_ready.record(getCurrentCUDAStream(dst_device.index()));

    device_guard.set_device(src_device);
    dst_ready.block(copy_stream);
  }

  if (memcpy_eligible) {
    void *dst = iter.data_ptr(0);
    void *src = iter.data_ptr(1);
    size_t size = numel * iter.element_size(0);
    if (src != dst || src_device != dst_device) {
      // Due to bizarre cuda driver intricacies, copies of
      // cudaMallocAsynced memory between devices that aren't
      // peer-to-peer-capable need "hipMemcpyPeerAsync".
      // So we let the allocator implement the correct call
      // (either hipMemcpyAsync or hipMemcpyPeerAsync)
      AT_CUDA_CHECK(CUDACachingAllocator::memcpyAsync(
        dst, dst_device.index(),
        src, src_device.index(),
        size, copy_stream, p2p_enabled));
    }
  } else {
    if (same_neg) {
      if (!same_conj) {
        conj_kernel_cuda(iter);
      } else {
        direct_copy_kernel_cuda(iter);
      }
    } else {
      if (!same_conj) {
        neg_conj_kernel_cuda(iter);
      } else {
        neg_kernel_cuda(iter);
      }
    }
  }

  if (src_device != dst_device) {
    // dst waits on src barrier (dst already waits on dst). We cannot
    // operate on dst's copy until the copy is complete.

    // Still on src_device, record stream event
    CUDAEvent src_ready;
    src_ready.record(copy_stream);

    device_guard.set_device(dst_device);
    src_ready.block(getCurrentCUDAStream(dst_device.index()));
  }

  AT_CUDA_CHECK(hipGetLastError());
}

static bool copy_requires_temporaries(TensorIterator& iter, bool p2p_enabled) {
  Device dst_device = iter.device(0);
  Device src_device = iter.device(1);

  if (dst_device == src_device) {
    // We never require temporaries for copies on the same GPU.
    TORCH_INTERNAL_ASSERT(dst_device.is_cuda() && src_device.is_cuda());
    return false;
  }

  bool same_dtype = iter.dtype(0) == iter.dtype(1);
  if (same_dtype && iter.is_contiguous()) {
    // Contiguous same-dtype copies can always use hipMemcpyAsync
    return false;
  } else if (dst_device.is_cuda() && src_device.is_cuda()) {
    // Copies between GPUs can use the copy kernel if P2P is supported
    return !p2p_enabled;
  } else {
    // The remaining cases require temporaries. For example, this includes
    // non-contiguous copies between CPU and GPU.
    return true;
  }
}

static bool maybe_enable_p2p_access(Device dst_device, Device src_device) {
  if (dst_device.is_cpu() || src_device.is_cpu()) {
    return false;
  }
  return at::cuda::get_p2p_access(src_device.index(), dst_device.index());
}

static void copy_kernel_cuda(TensorIterator& iter, bool non_blocking) {
  TORCH_CHECK(iter.ntensors() == 2);

  Device dst_device = iter.device(0);
  Device src_device = iter.device(1);

  // Enable p2p access between devices. (No-op if it involves the CPU)
  bool p2p_enabled = maybe_enable_p2p_access(dst_device, src_device);

  if (copy_requires_temporaries(iter, p2p_enabled)) {
    // NB: this involves recursive calls to copy. Be careful that those copies
    // don't require temporaries or you will cause an infinite recursion!
    auto& dst = iter.tensor(0);
    Tensor dst_contig;
    Tensor src_contig;

    // If non_blocking is true - type conversions are performed on the GPU
    // For blocking transfers conversions are performed on CPU to avoid allocating
    // extra GPU memory
    // for GPU-GPU transfers conversions are performed on the source device
    auto conversion_device = non_blocking ? kCUDA : kCPU;
    if (iter.device_type(1) == conversion_device) {
      dst_contig = dst.is_contiguous() ? dst : at::empty_like(dst, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
      src_contig = iter.tensor(1).to(iter.dtype(0)).expand_as(dst).contiguous();
    } else {
      bool same_type = iter.dtype(0) == iter.dtype(1);
      dst_contig = (dst.is_contiguous() && same_type) ? dst : at::empty_like(dst, iter.dtype(1), LEGACY_CONTIGUOUS_MEMORY_FORMAT);
      src_contig = iter.tensor(1).expand_as(dst).contiguous();
    }

    // propagate the correct conjugate bit
    dst_contig._set_conj(dst.is_conj());
    src_contig._set_conj(iter.tensor(1).is_conj());

    dst_contig._set_neg(dst.is_neg());
    src_contig._set_neg(iter.tensor(1).is_neg());

    // perform a same-dtype copy on contiguous tensors
    TORCH_INTERNAL_ASSERT(dst_contig.sizes().equals(src_contig.sizes()));
    TORCH_INTERNAL_ASSERT(dst_contig.scalar_type() == src_contig.scalar_type());
    dst_contig.copy_(src_contig, non_blocking);

    // if necessary, copy back into dst
    if (!dst_contig.is_same(dst)) {
      TORCH_INTERNAL_ASSERT(dst_contig.device() == dst.device());
      dst.copy_(dst_contig, non_blocking);
    }
    return;
  }

  // Copy on GPU (or between GPUs)
  if (dst_device.is_cuda() && src_device.is_cuda()) {
    copy_device_to_device(iter, non_blocking, p2p_enabled);
    return;
  }

  // Copy between CPU and GPU
  cuda::OptionalCUDAGuard device_guard;
  hipMemcpyKind kind;
  if (dst_device.is_cuda() && src_device.is_cpu()) {
    device_guard.set_device(dst_device);
    kind = hipMemcpyHostToDevice;
  } else if (dst_device.is_cpu() && src_device.is_cuda()) {
    device_guard.set_device(src_device);
    kind = hipMemcpyDeviceToHost;
  } else {
    TORCH_INTERNAL_ASSERT(false, "unsupported devices in GPU copy_()");
  }

  void* dst = iter.data_ptr(0);
  void* src = iter.data_ptr(1);
  int64_t nbytes = iter.numel() * iter.element_size(0);
  CUDAStream stream = getCurrentCUDAStream();

  if (non_blocking) {
    AT_CUDA_CHECK(hipMemcpyAsync(dst, src, nbytes, kind, stream));
    // we use both the storage context and the tensor data pointer as the key
    // for the caching host allocator. This allows us to better attribute the
    // events to the original tensor allocation correctly. The cases we seek to
    // handle are:

    // 1: a user can pass a pinned memory tensor with an alternative
    // context, for example if allocating memory directly from the pinned memory
    // allocator and constructing a tensor with torch::from_blob.

    // 2: a user can pass a tensor with a different base pointer to the original
    // allocation (via slicing).
    const auto& dst_tensor = iter.tensor(0);
    const auto& src_tensor = iter.tensor(1);
    const auto& host_tensor = (dst_device == kCPU ? dst_tensor : src_tensor);
    auto* ptr = (dst_device == kCPU ? dst : src);
    auto* ctx = host_tensor.storage().data_ptr().get_context();
    // TODO: warn on the return value.
    CachingHostAllocator_recordEvent(ptr, ctx, stream);

  } else {
    at::cuda::memcpy_and_sync(dst, src, nbytes, kind, stream);
  }

  if (iter.tensor(0).is_conj() != iter.tensor(1).is_conj()) {
     iter.tensor(0).conj_physical_();
  }
  if (iter.tensor(0).is_neg() != iter.tensor(1).is_neg()) {
     iter.tensor(0).neg_();
  }
}

REGISTER_DISPATCH(copy_stub, &copy_kernel_cuda)

} // namespace at::native
